#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Matrix.h"

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
      {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}



const short N = 4 ;

// CUDA Kernel for Matrix Addition
__global__ void Matrix_Addition ( const int *dev_a , const int *dev_b , int *dev_c)
{
      //Get the id of thread within a block
      unsigned short tid = threadIdx.x ;
     
      if ( tid < N ) // check the boundry condition for the threads
            dev_c [tid] = dev_a[tid] + dev_b[tid] ;

}


int main (void)
{
    int Host_a[N], Host_b[N], Host_c[N];
    //Device array
    int *dev_a , *dev_b, *dev_c ;
    //Allocate the memory on the GPU
    HANDLE_ERROR ( hipMalloc((void **)&dev_a , N*sizeof(int) ) );
    HANDLE_ERROR ( hipMalloc((void **)&dev_b , N*sizeof(int) ) );
    HANDLE_ERROR ( hipMalloc((void **)&dev_c , N*sizeof(int) ) );

    //fill the Host array with random elements on the CPU
    Matrix<int> a(2, 2);
    Matrix<int> b(2, 2);
    a = {2,2,2,2};
    b = {1, 2, 3, 4};
    //cout<<a;
    //Host_a = a.matrToArr();
    //Host_b = b.matrToArr();
    int el=0;

    for (int i = 0; i < 2 ; i++ )
    {
        for(int j = 0; j< 2; j++) {
            Host_a[el] = a(i, j);
            Host_b[el] = b(i, j);
            el++;
        }
    }
   // cout<< el<< endl;

    //Copy Host array to Device array
    HANDLE_ERROR (hipMemcpy (dev_a , Host_a , N*sizeof(int) , hipMemcpyHostToDevice));
    HANDLE_ERROR (hipMemcpy (dev_b , Host_b , N*sizeof(int) , hipMemcpyHostToDevice));

    //Make a call to GPU kernel
    Matrix_Addition <<< 1, N  >>> (dev_a , dev_b , dev_c ) ;

    //Copy back to Host array from Device array
    HANDLE_ERROR (hipMemcpy(Host_c , dev_c , N*sizeof(int) , hipMemcpyDeviceToHost));

    for ( int i = 0; i<N; i++ )
        printf ("%d\n", Host_c[i] ) ;

    //Free the Device array memory
    hipFree (dev_a) ;
    hipFree (dev_b) ;
    hipFree (dev_c) ;

    system("pause");
    return 0 ;

}
